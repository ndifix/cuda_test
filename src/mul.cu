
#include <hip/hip_runtime.h>

__global__ void
mul2_cuda_impl (
    int *xs,
    const size_t n
    )
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        xs[i] *= 2;
    }
}

extern "C" void
mul2_cuda (
    int *xs,
    const size_t n
    )
{
    size_t bytes = n * sizeof(int);

    int *dev_xs;
    hipMalloc(&dev_xs, bytes);
    hipMemcpy(dev_xs, xs, bytes, hipMemcpyHostToDevice);

    dim3 block(1024);
    dim3 grid((n + block.x - 1) / block.x);
    mul2_cuda_impl<<<grid, block>>>(dev_xs, n);

    hipMemcpy(xs, dev_xs, bytes, hipMemcpyDeviceToHost);
    hipFree(dev_xs);
    hipDeviceReset();
}

extern "C" void
mul2_cuda_copy (
    const int *xs,
    int *ys,
    const size_t n
    )
{
    size_t bytes = n * sizeof(int);

    int *dev_xs;
    hipMalloc(&dev_xs, bytes);
    hipMemcpy(dev_xs, xs, bytes, hipMemcpyHostToDevice);

    dim3 block(1024);
    dim3 grid((n + block.x - 1) / block.x);
    mul2_cuda_impl<<<grid, block>>>(dev_xs, n);

    hipMemcpy(ys, dev_xs, bytes, hipMemcpyDeviceToHost);
    hipFree(dev_xs);
    hipDeviceReset();
}
