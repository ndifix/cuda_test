
#include <hip/hip_runtime.h>

__global__ void
mul2_cuda_impl (
    int *xs,
    const size_t n
    )
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        xs[i] *= 2;
    }
}

extern "C" void
mul2_cuda (
    int *xs,
    const size_t n
    )
{
    size_t bytes = n * sizeof(int);

    int *dev_xs;
    hipMalloc(&dev_xs, bytes);
    hipMemcpy(dev_xs, xs, bytes, hipMemcpyHostToDevice);

    dim3 block(1024);
    dim3 grid((n + block.x - 1) / block.x);
    mul2_cuda_impl<<<grid, block>>>(dev_xs, n);

    hipMemcpy(xs, dev_xs, bytes, hipMemcpyDeviceToHost);
    hipFree(dev_xs);
    hipDeviceReset();
}

extern "C" void
mul2_cuda_copy (
    const int *xs,
    int *ys,
    const size_t n
    )
{
    size_t bytes = n * sizeof(int);

    int *dev_xs;
    hipMalloc(&dev_xs, bytes);
    hipMemcpy(dev_xs, xs, bytes, hipMemcpyHostToDevice);

    dim3 block(1024);
    dim3 grid((n + block.x - 1) / block.x);
    mul2_cuda_impl<<<grid, block>>>(dev_xs, n);

    hipMemcpy(ys, dev_xs, bytes, hipMemcpyDeviceToHost);
    hipFree(dev_xs);
    hipDeviceReset();
}

__global__ void
mul_cuda_impl (
    int *a,
    int *b,
    int *c,
    const size_t n
    )
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] * b[i];
    }
}

extern "C" void
mul_cuda (
    int *as,
    int *bs,
    int *cs,
    const size_t n
    )
{
    size_t bytes = n * sizeof(int);

    int *dev_as, *dev_bs, *dev_cs;
    hipMalloc(&dev_as, bytes);
    hipMalloc(&dev_bs, bytes);
    hipMalloc(&dev_cs, bytes);
    hipMemcpy(dev_as, as, bytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_bs, bs, bytes, hipMemcpyHostToDevice);

    dim3 block(1024);
    dim3 grid((n + block.x - 1) / block.x);
    mul_cuda_impl<<<grid, block>>>(dev_as, dev_bs, dev_cs, n);

    hipMemcpy(cs, dev_cs, bytes, hipMemcpyDeviceToHost);
    hipFree(dev_as);
    hipFree(dev_bs);
    hipFree(dev_cs);
    hipDeviceReset();
}
